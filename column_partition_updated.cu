
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel for column partitioning
__global__ void columnPartitionKernel(int *matrix, int numRows, int numCols, int partitionSize, int *result) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows) {
        int quotientSize = numCols / partitionSize;
        int *binaryVector = result + row * quotientSize;

        // Initialize the binary vector with zeros
        for (int j = 0; j < quotientSize; j++) {
            binaryVector[j] = 0;
        }

        // Check if there is a non-zero element in the current block in the partition
        for (int j = 0; j < numCols; j++) {
            int blockIndex = j / partitionSize; // Determine the block index

            // Check if there is a non-zero element in the current block in the partition
            if (matrix[row * numCols + j] != 0) {
                binaryVector[blockIndex] = 1;
            }
        }
    }
}

// Function to write a result matrix to a text file for GPU
void writeResultMatrixToFileGPU(int *result, int numRows, int quotientSize, const char *filename) {
    FILE *file = fopen(filename, "w");

    if (file == NULL) {
        perror("Error opening file");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < quotientSize; j++) {
            fprintf(file, "%d ", result[i * quotientSize + j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);
}

// Function to perform column partitioning on GPU
void columnPartition(int *matrix, int numRows, int numCols, int partitionSize) {
    int quotientSize = numCols / partitionSize;

    // Allocate GPU memory for matrix and result
    int *d_matrix, *d_result;
    hipMalloc((void **)&d_matrix, numRows * numCols * sizeof(int));
    hipMemcpy(d_matrix, matrix, numRows * numCols * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_result, numRows * quotientSize * sizeof(int));

    // Launch the kernel with appropriate block and grid dimensions
    int blockSize = 256;
    int gridSize = (numRows + blockSize - 1) / blockSize;
    columnPartitionKernel<<<gridSize, blockSize>>>(d_matrix, numRows, numCols, partitionSize, d_result);

    // Copy the result back from GPU to CPU
    int *result = (int *)malloc(numRows * quotientSize * sizeof(int));
    hipMemcpy(result, d_result, numRows * quotientSize * sizeof(int), hipMemcpyDeviceToHost);

    // Print the binary vectors for each row
    for (int i = 0; i < numRows; i++) {
        printf("Binary Vector for Row %d: ", i + 1);
        for (int j = 0; j < quotientSize; j++) {
            printf("%d ", result[i * quotientSize + j]);
        }
        printf("\n");
    }

    // Write the result matrix to a text file
    writeResultMatrixToFileGPU(result, numRows, quotientSize, "output_matrix_gpu.txt");

    // Free GPU memory
    hipFree(d_matrix);
    hipFree(d_result);
    free(result);
}

// Function to generate a random sparse matrix
void generateRandomSparseMatrix(int *matrix, int numRows, int numCols, double sparsity) {
    srand(42); // Seed the random number generator

    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numCols; j++) {
            // Generate a random number between 0 and 1
            double randomValue = (double)rand() / RAND_MAX;

            // Check if the random value is less than the desired sparsity
            if (randomValue < sparsity) {
                // Set a non-zero value
                matrix[i * numCols + j] = rand() % 10 + 1; // Values between 1 and 10
            } else {
                // Set a zero value
                matrix[i * numCols + j] = 0;
            }
        }
    }
}

int main() {
    // Example matrix dimensions
    int numRows = 100;
    int numCols = 100;
        // Allocate memory for the matrix
    int *matrix = (int *)malloc(numRows * numCols * sizeof(int));
    
    // Generate a random matrix
    generateRandomSparseMatrix(matrix, numRows, numCols, 0.25); // 25% sparse matrix

    // Perform column partitioning on GPU
    columnPartition((int *)matrix, numRows, numCols, 2);

    return 0;
}
