#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include "merging.h"
#include <stdio.h>
// Write a vector to file
extern "C"

void computeGold(unsigned int* outVector, unsigned int* inVector,
                unsigned int* blocks_height, unsigned int* outPattern,
                unsigned int* group_indices_data, unsigned int* group_indices_csr,
                float tau, unsigned int blocks_across_width, unsigned int height,int* grp_count, unsigned int* order_ds, int* group,  unsigned int* NNZ_pattern);
void WriteFile(unsigned int* Vector, char* file_name, unsigned int width, int height);
void WriteFile_2(int* Vector, char* file_name, unsigned int width, int height);
void Initialize(unsigned int* Vector, unsigned int width, unsigned int height, unsigned int val);
void MergeOnDevice(Out_1DSAAD out, In_1DSAAD in, float tau, unsigned int blocks_across_width, unsigned int height, unsigned int* group_indices_data, unsigned int* group_indices_csr);
void startTime(Timer* timer);
void stopTime(Timer* timer);
float elapsedTime(Timer timer);
int ReadFile(unsigned int* Vector, char* file_name, unsigned int width, int height);
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
    
    float tau = atof(argv[1]);
    unsigned int blocks_across_width = atoi(argv[2]);
    unsigned int height = atoi(argv[3]);
    unsigned int* inVector = ( unsigned int*) malloc(height*blocks_across_width*sizeof(unsigned int));
    unsigned int* outVector = ( unsigned int*) malloc(height*blocks_across_width*sizeof(unsigned int));
    int* group =(int*) malloc(height*sizeof(int));
    unsigned int* blocks_height =( unsigned int*) malloc(height*sizeof(unsigned int));
    unsigned int* outPattern = (unsigned int*) malloc(height*blocks_across_width*sizeof(unsigned int));
    //for(unsigned int i = 0; i < height * blocks_across_width; i++)
	//{
		//inVector[i]=1;
        //inVector[i] = rand()%2;
        //if(i<height){
            //group[i]=-1;
        //}
	//}
    //inVector[i]=1;
    for(unsigned int i = 0; i < height ; i++){
        group[i]=-1;
    }
    unsigned int* group_indices_data =( unsigned int*) malloc(height*sizeof(unsigned int));;
    unsigned int* group_indices_csr =( unsigned int*) malloc(blocks_across_width*sizeof(unsigned int));;
    char* file_name[3] = {"./input.txt", "./row_idx.txt", "./grp_ptr.txt"};
    ReadFile(inVector, file_name[0], blocks_across_width, height);
    ReadFile(group_indices_data, file_name[1], 1, height);
    ReadFile(group_indices_csr, file_name[2], 1, height);
    //unsigned int group_indices_data[8]={0,2,5,7,1,3,6,4};
    //unsigned int group_indices_csr[5]={0,0,4,6,7};
    int* group_count=(int*) malloc(2*sizeof(int));
    printf("tau, blocks_across_width, height, %f %d %d\n ", tau, blocks_across_width, height);
    //char* file_name = argv[4];
    //WriteFile(inVector, file_name, blocks_across_width, height);
    unsigned int* order_ds = (unsigned int*) malloc(height*sizeof(unsigned int));
    unsigned int* NNZ_pattern = (unsigned int*) malloc(height*sizeof(unsigned int));
   
    Initialize(NNZ_pattern, 1, height, 0);
    Initialize(blocks_height, 1, height, 1);

     //GPU
    Out_1DSAAD out_d;
    In_1DSAAD in_d;
	hipMalloc((void**)&in_d.group, height*sizeof(int));
	hipMalloc((void**)&out_d.outVector, height*blocks_across_width*sizeof(unsigned int));
	hipMalloc((void**)&in_d.inVector, height*blocks_across_width*sizeof(unsigned int));
	hipMalloc((void**)&out_d.blocks_height, height*sizeof(unsigned int));
	hipMalloc((void**)&out_d.pattern_indices, height*sizeof(unsigned int));
	hipMalloc((void**)&out_d.outPattern, height*blocks_across_width*sizeof(unsigned int));
	hipMalloc((void**)&out_d.midPattern, height*blocks_across_width*sizeof(unsigned int));
	hipMalloc((void**)&in_d.group_indices_data, height*sizeof(unsigned int));
	hipMalloc((void**)&in_d.group_indices_csr, (blocks_across_width+1)*sizeof(unsigned int));
    
    hipMemcpy(in_d.group, group, height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(in_d.inVector, inVector, height*blocks_across_width*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(in_d.group_indices_data, group_indices_data, height*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(in_d.group_indices_csr, group_indices_csr, (blocks_across_width+1)*sizeof(unsigned int), hipMemcpyHostToDevice);

    Timer timer;
    hipError_t cuda_ret;
    startTime(&timer);
    computeGold(outVector, inVector, blocks_height, outPattern, group_indices_data, group_indices_csr, tau, blocks_across_width, height, group_count,order_ds, group, NNZ_pattern);
    stopTime(&timer); printf("CPU Time: %f s\n", elapsedTime(timer));
    char* file_name_param[4] ={"./Data/order_ds.txt", "./Data/group.txt","./Data/blocks_height.txt", "./Data/outPattern.txt"};
    WriteFile(order_ds, file_name_param[0], 1,group_count[1]);
    WriteFile_2(group, file_name_param[1], 1,height);
    WriteFile(blocks_height, file_name_param[2], 1,group_count[0]);
    WriteFile(outPattern, file_name_param[3], blocks_across_width, group_count[0]);
    char* file_name_out ="./Data/outVector.txt";
    WriteFile(outVector, file_name_out, blocks_across_width, group_count[1]);

    free(inVector);
    
    startTime(&timer);
    MergeOnDevice(out_d, in_d, tau, blocks_across_width, height, group_indices_data, group_indices_csr);
    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("GPU Time: %f s\n", elapsedTime(timer));

    hipFree(in_d.inVector);hipFree(in_d.group_indices_data);hipFree(in_d.group_indices_csr);
    unsigned int* outPattern_gpu = ( unsigned int*) malloc(height*blocks_across_width*sizeof(unsigned int));
    unsigned int* outVector_gpu = ( unsigned int*) malloc(height*blocks_across_width*sizeof(unsigned int));
    unsigned int group_count_gpu;
    int* group_gpu =(int*) malloc(height*sizeof(int));
    cuda_ret = hipMemcpy(outVector_gpu, out_d.outVector, sizeof(unsigned int)*blocks_across_width*height, 
        hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory from device");
    cuda_ret = hipMemcpy(outPattern_gpu, out_d.outPattern, sizeof(unsigned int)*blocks_across_width*height, 
        hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory from device");
    cuda_ret = hipMemcpy(group_gpu, in_d.group, sizeof(int)*height, 
        hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory from device");
    cuda_ret = hipMemcpy(&group_count_gpu, out_d.pattern_indices+height-1, sizeof(unsigned int), 
        hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory from device");
    char* file_name_param_gpu[3] ={"./Data/outVector_gpu.txt", "./Data/group_gpu.txt", "./Data/outPattern_gpu.txt"};
    WriteFile(outVector_gpu+blocks_across_width, file_name_param_gpu[0], blocks_across_width,height-1);
    WriteFile(outPattern_gpu+blocks_across_width, file_name_param_gpu[2], blocks_across_width,group_count_gpu);
    WriteFile_2(group_gpu, file_name_param_gpu[1], 1,height);

    //compare outVector, outPattern, group;
    hipFree(out_d.outVector); hipFree(out_d.blocks_height); hipFree(out_d.outPattern);hipFree(in_d.group);
    free(outVector), free(blocks_height); free(outPattern);free(outVector_gpu);free(outPattern_gpu);free(group_gpu);
}

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

